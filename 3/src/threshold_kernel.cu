#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <numeric>

#include "threshold_cuda.hpp"


namespace {

constexpr auto min_value = std::numeric_limits<std::uint8_t>::min();
constexpr auto max_value = std::numeric_limits<std::uint8_t>::max();

__global__ void threshold_kernel(const std::uint8_t* const src, std::uint8_t* const dst,
                                 const std::size_t width, const std::size_t height,
                                 const std::uint8_t thresh) {
    const auto idx = blockDim.x * blockIdx.x + threadIdx.x;

    dst[idx] = src[idx] < thresh ? min_value : max_value;
}

}  // anonymous namespace


namespace cuda {

void threshold(const std::uint8_t* const src, std::uint8_t* const dst,
               const std::size_t width, const std::size_t height, const std::uint8_t thresh) {
    threshold_kernel<<<height, width>>>(src, dst, width, height, thresh);
}

}  // namespace cuda
